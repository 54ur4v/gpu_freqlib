#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>

#include <gpu_freqlib/freqlib.h>

int main() {
  freqlib::instance knob;

  printf("Current SM clock: %u\n",
         knob.get_current_clock());
  printf("Current memory clock: %u\n",
         knob.get_current_mem_clock());

  auto sm_clocks = knob.get_supported_clocks();
  auto mem_clocks = knob.get_supported_mem_clocks();

  //knob.step_up_mem_clock();
  knob.step_down_clock();
  knob.step_down_clock();

  printf("Current SM clock: %u\n",
         knob.get_current_clock());

  //std::vector<double> clock_frequencies{knob.get_supported_clocks()};
  //std::vector<double> mem_frequencies{knob.get_supported_mem_clocks()};

  return 0;
}

#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>

#include <gpu_freqlib/freqlib.h>

int main() {
  freqlib::instance knob;

  printf("Current SM clock: %u\n",
         knob.get_current_clock());
  printf("Current memory clock: %u\n",
         knob.get_current_mem_clock());

  auto sm_clocks = knob.get_supported_clocks();
  auto mem_clocks = knob.get_supported_mem_clocks();

  //std::vector<double> clock_frequencies{knob.get_supported_clocks()};
  //std::vector<double> mem_frequencies{knob.get_supported_mem_clocks()};

  return 0;
}
